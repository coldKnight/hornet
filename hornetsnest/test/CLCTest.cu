/**
 * @brief Breadth-first Search Top-Down test program
 * @file
 */
#include "Static/ClosenessCentrality/clc.cuh"
#include "Static/ClosenessCentrality/exact_clc.cuh"
#include "Static/ClosenessCentrality/approximate_clc.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

using namespace std;
using namespace graph;
using namespace graph::structure_prop;
using namespace graph::parsing_prop;

int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;

    // GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    // graph::GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv,false);
    Timer<DEVICE> TM;


    // graph.read(argv[1], SORT | PRINT_INFO);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);

    ClosenessCentrality clc(hornet_graph);

	vid_t root = graph.max_out_degree_id();
	if (argc==3)
	  root = atoi(argv[2]);
    // root = 226410;
    cout << "Root is " << root << endl;
    clc.reset();
    clc.setRoot(root);

    hipProfilerStart();TM.start();
    clc.run();

    TM.stop();hipProfilerStop();
    TM.print("ClosenessCentrality");
#if 0

    ExactCLC eclc(hornet_graph);

    eclc.reset();

    hipProfilerStart();TM.start();
    // eclc.run();
    TM.stop();hipProfilerStop();
    TM.print("Exact ClosenessCentrality");

    vid_t numRoots=1000;
    vid_t* roots = new vid_t[numRoots];
    ApproximateCLC::generateRandomRootsUniform(hornet_graph.nV(), numRoots, &roots, 1 );

    ApproximateCLC aclc(hornet_graph, roots,numRoots);
    aclc.reset();

    hipProfilerStart();TM.start();
    // aclc.run();
    TM.stop();hipProfilerStop();
    TM.print("Approximate ClosenessCentrality");


    delete[] roots;
#endif
    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
#if defined(RMM_WRAPPER)
    hornets_nest::gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
#endif

    ret = exec(argc, argv);

#if defined(RMM_WRAPPER)
    }//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    hornets_nest::gpu::finalizeRMMPoolAllocation();
#endif

    return ret;
}

